#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>
#include <cmath>

#include <hip/hip_runtime.h>
#include <cassert>

#include "test/py_interface/py_interface_cuda.cuh"

#define CHECK_CUDA(err) ({ \
  if (err != hipSuccess) { \
    auto err_str = hipGetErrorString(err); \
    std::cout << __FILE__ << ":" << __LINE__ << " @ " << __func__ << ": CUDA Failed with (err=" << err << "): " << err_str << std::endl; \
    assert(err == hipSuccess); \
  } \
})

using clock_value_t = long long;

using steady_clock = std::chrono::steady_clock;

__global__ void _gpu_sleep(clock_value_t sleep_cycles, int64_t* output)
{
  clock_value_t start = clock64();
  clock_value_t cycles_elapsed;
  do {
    cycles_elapsed = clock64() - start;
    *output = *output + 1;
  }
  while (cycles_elapsed < sleep_cycles);
}

double GPUClockFreq::gpu_sleep(clock_value_t sleep_cycles) {
//  int64_t output = 0;
  int64_t* output = nullptr;
  hipError_t err;
  err = hipHostMalloc((void**)&output, sizeof(int64_t));
  CHECK_CUDA(err);
  *output = 0;

  auto start_t = GPUClockFreq::now();
  _gpu_sleep<<<1, 1>>>(sleep_cycles, output); // This line alone is 0.208557334
  err = hipDeviceSynchronize();
  CHECK_CUDA(err);
  auto end_t = GPUClockFreq::now(); // This whole block is 5.316381218, but we measure it using nvprof as 5.113029

//  std::cout << "> gpu_sleep.output=" << *output << ", sleep_cycles=" << sleep_cycles << std::endl;

  err = hipHostFree(output);
  CHECK_CUDA(err);

  auto time_sec = GPUClockFreq::elapsed_sec(start_t, end_t);
  return time_sec;
}

void GPUClockFreq::guess_cycles() {
  std::cout << "> Using initial sleep_cycles=" << _sleep_cycles << std::endl;
  while (true) {
    time_type start_t, end_t;
    iter(&start_t, &end_t);
    auto total_sec = elapsed_sec(start_t, end_t);
    if (total_sec > GPU_CLOCK_MIN_SAMPLE_TIME_SEC) {
      std::cout << "> Using sleep_cycles=" << _sleep_cycles << ", which takes " << total_sec << " seconds" << std::endl;
      break;
    } else if (total_sec > GPU_CLOCK_MIN_GUESS_TIME_SEC) {
      double cycles_per_second = _sleep_cycles / total_sec;
      auto seconds_left = GPU_CLOCK_MIN_SAMPLE_TIME_SEC - total_sec;
      // Add fudge factor of 10% more than we believe we need.
      auto guess_cycles_left = 1.1 * seconds_left * cycles_per_second;
      auto new_sleep_cycles = _sleep_cycles + guess_cycles_left;
      if (!(new_sleep_cycles > _sleep_cycles)) {
        std::cout << "total_sec = " << total_sec
                  << ", new_sleep_cycles = " << new_sleep_cycles
                  << ",  _sleep_cycles = " << _sleep_cycles
                  << std::endl;
        assert(new_sleep_cycles > _sleep_cycles);
      }
      _sleep_cycles = new_sleep_cycles;
    } else {
      auto new_sleep_cycles = _sleep_cycles * 2;
      if (!(new_sleep_cycles > _sleep_cycles)) {
        std::cout << "total_sec = " << total_sec
                  << ", new_sleep_cycles = " << new_sleep_cycles
                  << ",  _sleep_cycles = " << _sleep_cycles
                  << std::endl;
        assert(new_sleep_cycles > _sleep_cycles);
      }
      _sleep_cycles = new_sleep_cycles;
    }
  }
}

GPUClockFreq::time_type GPUClockFreq::now() {
  time_type t = steady_clock::now();
  return t;
}

double GPUClockFreq::elapsed_sec(time_type start, time_type stop) {
  double sec = ((stop - start).count()) * steady_clock::period::num / static_cast<double>(steady_clock::period::den);
  return sec;
}

void GPUClockFreq::iter(time_type *start_t, time_type *end_t) {
  *start_t = now();
  GPUClockFreq::gpu_sleep(_sleep_cycles);
  *end_t = now();
}

double GPUClockFreq::freq_mhz(double time_sec) {
  return _sleep_cycles / time_sec;
}

GPUClockResult GPUClockFreq::run() {
  guess_cycles();
  time_type start_t, end_t;
  for (int r = 0; r < _repetitions; ++r) {
    iter(&start_t, &end_t);
    auto total_sec = elapsed_sec(start_t, end_t);
    _time_secs.push_back(total_sec);
    auto freq = freq_mhz(total_sec);
    _freq_mhz.push_back(freq);
    std::cout << "> freq[" << r << "] = " << freq << " MHz" << std::endl;
  }

  auto avg = Average(_freq_mhz);
  auto std = Std(_freq_mhz);
  _result = GPUClockResult{.avg_mhz = avg, .std_mhz = std};
  std::cout << "> Average freq = " << avg << " MHz" << std::endl;
  std::cout << "> Std freq = " << std << " MHz" << std::endl;

  return _result;
}

