#include "hip/hip_runtime.h"
//
// Created by jgleeson on 2020-05-14.
//

#include "CommonCuda.cuh"

#include <iostream>

namespace CuptiSamples {

// Device code
__global__ void VecAdd(const int* A, const int* B, int* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

// Device code
__global__ void VecSub(const int* A, const int* B, int* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] - B[i];
}

static void initVec(int *vec, int n)
{
    for (int i = 0; i < n; i++)
        vec[i] = i;
}

void ComputeVecAdd(int iters, size_t n_int32s)
{
    size_t size = n_int32s * sizeof(int);
    int threadsPerBlock = 0;
    int blocksPerGrid = 0;
    int sum;
    int *h_A, *h_B, *h_C;
    int *d_A, *d_B, *d_C;

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);

    // Initialize input vectors
    initVec(h_A, n_int32s);
    initVec(h_B, n_int32s);
    memset(h_C, 0, size);

    // Allocate vectors in device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel (multiple times to make sure we have time for
    // sampling)
    threadsPerBlock = 256;
    blocksPerGrid = (n_int32s + threadsPerBlock - 1) / threadsPerBlock;
    for (int i = 0; i < iters; i++) {
        VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n_int32s);
    }


    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (size_t i = 0; i < n_int32s; ++i) {
        sum = h_A[i] + h_B[i];
        if (h_C[i] != sum) {
            std::cout << "kernel execution FAILED" << std::endl;
            exit(-1);
        }
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
}

void ComputeVectorAddSubtract(size_t N)
{
    // int N = 50000;
    size_t size = N * sizeof(int);
    int threadsPerBlock = 0;
    int blocksPerGrid = 0;
    int *h_A, *h_B, *h_C, *h_D;
    int *d_A, *d_B, *d_C, *d_D;
    int sum, diff;

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size);
    h_B = (int*)malloc(size);
    h_C = (int*)malloc(size);
    h_D = (int*)malloc(size);

    // Initialize input vectors
    initVec(h_A, N);
    initVec(h_B, N);
    memset(h_C, 0, size);
    memset(h_D, 0, size);

    // Allocate vectors in device memory
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);
    hipMalloc((void**)&d_D, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    threadsPerBlock = 256;
    blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    printf("Launching kernel: blocks %d, thread/block %d\n",
           blocksPerGrid, threadsPerBlock);

    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    VecSub<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_D, N);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipMemcpy(h_D, d_D, size, hipMemcpyDeviceToHost);

    // Verify result
    for (size_t i = 0; i < N; ++i) {
        sum = h_A[i] + h_B[i];
        diff = h_A[i] - h_B[i];
        if (h_C[i] != sum || h_D[i] != diff) {
            fprintf(stderr, "error: result verification failed\n");
            exit(-1);
        }
    }

    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);
    if (d_D)
        hipFree(d_D);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
    if (h_D)
        free(h_D);
}

} // namespace CuptiSamples
