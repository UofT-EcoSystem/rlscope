#include "hip/hip_runtime.h"
/*
 * Copyright 2010-2017 NVIDIA Corporation. All rights reserved
 *
 * Sample app to demonstrate use of CUPTI library to obtain device timestamps
 * using callbacks for CUDA runtime APIs
 *
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cupti.h>

#define CHECK_CU_ERROR(err, cufunc)                                     \
  if (err != hipSuccess)                                              \
    {                                                                   \
      printf ("%s:%d: error %d for CUDA Driver API function '%s'\n",    \
              __FILE__, __LINE__, err, cufunc);                         \
      exit(-1);                                                         \
    }

#define CHECK_CUPTI_ERROR(err, cuptifunc)                               \
  if (err != CUPTI_SUCCESS)                                             \
    {                                                                   \
      const char *errstr;                                               \
      cuptiGetResultString(err, &errstr);                               \
      printf ("%s:%d:Error %s for CUPTI API function '%s'.\n",          \
              __FILE__, __LINE__, errstr, cuptifunc);                   \
      exit(-1);                                                         \
    }

// Structure to hold data collected by callback
typedef struct RuntimeApiTrace_st {
  const char *functionName;
  uint64_t startTimestamp;
  uint64_t endTimestamp;
  size_t memcpy_bytes;
  enum hipMemcpyKind memcpy_kind;
} RuntimeApiTrace_t;

enum launchOrder{ MEMCPY_H2D1, MEMCPY_H2D2, MEMCPY_D2H, KERNEL, THREAD_SYNC, LAUNCH_LAST};

// Vector addition kernel
__global__ void 
VecAdd(const int* A, const int* B, int* C, int N)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N)
    C[i] = A[i] + B[i];
}

// Initialize a vector
static void 
initVec(int *vec, int n)
{
  for (int i = 0; i < n; i++)
    vec[i] = i;
}

void CUPTIAPI
getTimestampCallback(void *userdata, CUpti_CallbackDomain domain,
                     CUpti_CallbackId cbid, const CUpti_CallbackData *cbInfo)
{
  static int memTransCount = 0;
  uint64_t startTimestamp;
  uint64_t endTimestamp;
  RuntimeApiTrace_t *traceData = (RuntimeApiTrace_t*)userdata;
  CUptiResult cuptiErr;
      
  // Data is collected only for the following API
  if ((cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020) ||
      (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000) ||
      (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaDeviceSynchronize_v3020) || 
      (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaMemcpy_v3020))  { 
     
    // Set pointer depending on API
    if ((cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020) ||
        (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000))
    {
      traceData = traceData + KERNEL;
    }
    else if (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaDeviceSynchronize_v3020) 
      traceData = traceData + THREAD_SYNC;
    else if (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaMemcpy_v3020)
      traceData = traceData + MEMCPY_H2D1 + memTransCount;
                 
    if (cbInfo->callbackSite == CUPTI_API_ENTER) {
      // for a kernel launch report the kernel name, otherwise use the API
      // function name.
      if (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunch_v3020 ||
          cbid == CUPTI_RUNTIME_TRACE_CBID_cudaLaunchKernel_v7000)
      {
        traceData->functionName = cbInfo->symbolName;
      }
      else {
        traceData->functionName = cbInfo->functionName;
      }

      // Store parameters passed to hipMemcpy
      if (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaMemcpy_v3020) {
        traceData->memcpy_bytes = ((cudaMemcpy_v3020_params *)(cbInfo->functionParams))->count;
        traceData->memcpy_kind = ((cudaMemcpy_v3020_params *)(cbInfo->functionParams))->kind;
      }
        
      // Collect timestamp for API start
      cuptiErr = cuptiDeviceGetTimestamp(cbInfo->context, &startTimestamp);
      CHECK_CUPTI_ERROR(cuptiErr, "cuptiDeviceGetTimestamp");
            
      traceData->startTimestamp = startTimestamp;
    }

    if (cbInfo->callbackSite == CUPTI_API_EXIT) {
      // Collect timestamp for API exit
      cuptiErr = cuptiDeviceGetTimestamp(cbInfo->context, &endTimestamp);
      CHECK_CUPTI_ERROR(cuptiErr, "cuptiDeviceGetTimestamp");
            
      traceData->endTimestamp = endTimestamp;
     
      // Advance to the next memory transfer operation
      if (cbid == CUPTI_RUNTIME_TRACE_CBID_cudaMemcpy_v3020) {
        memTransCount++;
      }
    } 
  }
}

static const char *
memcpyKindStr(enum hipMemcpyKind kind)
{
  switch (kind) {
  case hipMemcpyHostToDevice:
    return "HostToDevice";
  case hipMemcpyDeviceToHost:
    return "DeviceToHost";
  default:
    break;
  }

  return "<unknown>";
}

static void 
displayTimestamps(RuntimeApiTrace_t *trace)
{
  // Calculate timestamp of kernel based on timestamp from
  // hipDeviceSynchronize() call
  trace[KERNEL].endTimestamp = trace[THREAD_SYNC].endTimestamp;

  printf("startTimeStamp/gpuTime reported in nano-seconds\n\n");
  printf("Name\t\tStart Time\t\tGPU Time\tBytes\tKind\n");
  printf("%s\t%llu\t%llu\t\t%llu\t%s\n", trace[MEMCPY_H2D1].functionName,
         (unsigned long long)trace[MEMCPY_H2D1].startTimestamp, 
         (unsigned long long)trace[MEMCPY_H2D1].endTimestamp - trace[MEMCPY_H2D1].startTimestamp,
         (unsigned long long)trace[MEMCPY_H2D1].memcpy_bytes,
         memcpyKindStr(trace[MEMCPY_H2D1].memcpy_kind));
  printf("%s\t%llu\t%llu\t\t%llu\t%s\n", trace[MEMCPY_H2D2].functionName,
         (unsigned long long)trace[MEMCPY_H2D2].startTimestamp,
         (unsigned long long)trace[MEMCPY_H2D2].endTimestamp - trace[MEMCPY_H2D2].startTimestamp, 
         (unsigned long long)trace[MEMCPY_H2D2].memcpy_bytes,
         memcpyKindStr(trace[MEMCPY_H2D2].memcpy_kind)); 
  printf("%s\t%llu\t%llu\t\tNA\tNA\n", trace[KERNEL].functionName,
         (unsigned long long)trace[KERNEL].startTimestamp,
         (unsigned long long)trace[KERNEL].endTimestamp - trace[KERNEL].startTimestamp);
  printf("%s\t%llu\t%llu\t\t%llu\t%s\n", trace[MEMCPY_D2H].functionName,
         (unsigned long long)trace[MEMCPY_D2H].startTimestamp,
         (unsigned long long)trace[MEMCPY_D2H].endTimestamp - trace[MEMCPY_D2H].startTimestamp, 
         (unsigned long long)trace[MEMCPY_D2H].memcpy_bytes,
         memcpyKindStr(trace[MEMCPY_D2H].memcpy_kind)); 
}

static void 
cleanUp(int *h_A, int *h_B, int *h_C, int *d_A, int *d_B, int *d_C)
{
  if (d_A)
    hipFree(d_A);
  if (d_B)
    hipFree(d_B);
  if (d_C)
    hipFree(d_C);

  // Free host memory
  if (h_A)
    free(h_A);
  if (h_B)
    free(h_B);
  if (h_C)
    free(h_C);
}

int
main()
{
  hipCtx_t context = 0;
  hipDevice_t device = 0;
  hipError_t cuerr;
  CUptiResult cuptierr;
  int N = 50000;
  size_t size = N * sizeof(int);
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  int sum, i;
  int *h_A, *h_B, *h_C;
  int *d_A, *d_B, *d_C;

  CUpti_SubscriberHandle subscriber;
  RuntimeApiTrace_t trace[LAUNCH_LAST];
    
  cuerr = hipInit(0);
  CHECK_CU_ERROR(cuerr, "hipInit");

  cuerr = hipCtxCreate(&context, 0, device);
  CHECK_CU_ERROR(cuerr, "hipCtxCreate");

  cuptierr = cuptiSubscribe(&subscriber, (CUpti_CallbackFunc)getTimestampCallback , &trace);
  CHECK_CUPTI_ERROR(cuptierr, "cuptiSubscribe");

  cuptierr = cuptiEnableDomain(1, subscriber, CUPTI_CB_DOMAIN_RUNTIME_API);
  CHECK_CUPTI_ERROR(cuptierr, "cuptiEnableDomain");

  // Allocate input vectors h_A and h_B in host memory
  h_A = (int*)malloc(size);
  h_B = (int*)malloc(size);
  h_C = (int*)malloc(size);

  // Initialize input vectors
  initVec(h_A, N);
  initVec(h_B, N);
  memset(h_C, 0, size);

  // Allocate vectors in device memory
  hipMalloc((void**)&d_A, size);
  hipMalloc((void**)&d_B, size);
  hipMalloc((void**)&d_C, size);

  // Copy vectors from host memory to device memory
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  // Invoke kernel
  threadsPerBlock = 256;
  blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

  VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
  hipDeviceSynchronize();
    
  // Copy result from device memory to host memory
  // h_C contains the result in host memory
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    
  // Verify result
  for (i = 0; i < N; ++i) {
    sum = h_A[i] + h_B[i];
    if (h_C[i] != sum) {
      printf("kernel execution FAILED\n");
      goto Error;
    }
  }
 
  displayTimestamps(trace);

  cuptierr = cuptiUnsubscribe(subscriber);
  CHECK_CUPTI_ERROR(cuptierr, "cuptiUnsubscribe");

  cleanUp(h_A, h_B, h_C, d_A, d_B, d_C);
  hipDeviceSynchronize();
  return 0;

 Error:
  cleanUp(h_A, h_B, h_C, d_A, d_B, d_C);
  hipDeviceSynchronize();
  return -1;
}

